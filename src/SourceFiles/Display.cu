#include "hip/hip_runtime.h"
#include "../HeaderFiles/Display.cuh"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <math.h>
#include <stdio.h>
#include <fstream>
#include <chrono>
#include <thread>
#include "../../lib/SDL/include/SDL.h"
#include "../../lib/SDL/include/SDL_ttf.h"

#include "../HeaderFiles/Camera.h"
#include "../HeaderFiles/Button.h"


#define MAIN_MENU 0
#define RUNNING 1
#define PAUSED 2


__device__ float3 getVector(int x,int y,int imageWidth,int imageHeight,float cosX, float cosY, float sinX, float sinY,float fovScalar) {
    float heightToWidthRatio = static_cast<float>(imageHeight)/imageWidth;
    float3 rayVector = make_float3((static_cast<float>(2*x-imageWidth)/imageWidth),heightToWidthRatio*(static_cast<float>(imageHeight-2*y)/imageHeight),static_cast<float>(1.0*fovScalar));
    float inverseLength = rsqrtf(rayVector.x*rayVector.x+rayVector.y*rayVector.y+rayVector.z*rayVector.z);
    rayVector.x *= inverseLength;
    rayVector.y *= inverseLength;
    rayVector.z *= inverseLength;

    float verticalZ = rayVector.y*sinY + rayVector.z*cosY;
    return make_float3(rayVector.x*cosX+verticalZ*sinX,rayVector.y*cosY-rayVector.z*sinY,verticalZ*cosX-rayVector.x*sinX);
}


__device__ float3 getSkyBoxColor(float3 rayVector, float angleOfSun) {
    /* FUTURE IDEA:
     * Use an Image(s) of a SkyBox and In Python
     * Convert all of the RGB Values for each
     * pixels to floats between 0 and 1 and then
     * store that data in a file to be read in
     * for this program and then store that data in
     * an array and be prepared to convert from
     * a vector to a pixel from the skybox
    */
    float3 skyColor;
    float3 sunColor = make_float3(1.0f,1.0f,1.0f);
    if(angleOfSun > 0) {
        float angleBetweenRayAndSun = acos(rayVector.x*__cosf(angleOfSun) + rayVector.y*__sinf(angleOfSun));
        if(angleBetweenRayAndSun < 0.05) {
            skyColor.x = sunColor.x;
            skyColor.y = sunColor.y;
            skyColor.z = sunColor.z;
            return skyColor;
        }
        float t = 0.5f*(1.0f+(__sinf(acos(rsqrtf(rayVector.x*rayVector.x+rayVector.z*rayVector.z)*(rayVector.x*rayVector.x+rayVector.z*rayVector.z)))));
        float3 topSkyColor = make_float3(64.0f/255.0f,156.0f/255.0f,255.0f/255.0f);
        float3 bottomSkyColor = make_float3(1.0f,1.0f,1.0f);
        skyColor.x = topSkyColor.x*t + bottomSkyColor.x*(1-t);
        skyColor.y = topSkyColor.y*t + bottomSkyColor.y*(1-t);
        skyColor.z = topSkyColor.z*t + bottomSkyColor.z*(1-t);
        if(angleBetweenRayAndSun < 0.075) {
            int m = 1-10*fabs(angleBetweenRayAndSun-0.05);
            float colorTintX = min(1.0f,skyColor.x*m + sunColor.x*(1-m));
            float colorTintY = min(1.0f,skyColor.y*m + sunColor.y*(1-m));
            float colorTintZ = min(1.0f,skyColor.z*m + sunColor.z*(1-m));
            skyColor.x *= colorTintX;
            skyColor.y *= colorTintY;
            skyColor.z *= colorTintZ;

        }
        return skyColor;

    } else {
        skyColor.x = 0;
        skyColor.y = 0;
        skyColor.z = 0;
        return skyColor;
    }

}

__device__ float computeCrossProductLength(float3 vec1, float3 vec2) {
    return sqrtf(
        powf((vec1.y*vec2.z-vec2.y*vec1.z),2)+
        powf((vec1.z*vec2.x-vec2.z*vec1.x),2)+
        powf((vec1.x*vec2.y-vec2.x*vec1.y),2)
    );
}

__device__ float3 getRandomBounceVector(float3 normalVector, hiprandState* randomGenerator) {
    float3 randomBounceVector = make_float3(1.0f,0,0);
    float x;
    float y;
    float z;
    float squaredLength;
    for(int i = 0; i < 100; i ++) {
        x = hiprand_normal(randomGenerator);
        y = hiprand_normal(randomGenerator);
        z = hiprand_normal(randomGenerator);
        squaredLength = sqrtf(x*x+y*y+z*z); 
        if(squaredLength < 1) {
            randomBounceVector.x = x/squaredLength;
            randomBounceVector.y = y/squaredLength;
            randomBounceVector.z = z/squaredLength;
            break;
        }
    }
    if((normalVector.x*randomBounceVector.x + normalVector.y*randomBounceVector.y + normalVector.z*randomBounceVector.z) < 0) {
        randomBounceVector.x *= -1;
        randomBounceVector.y *= -1;
        randomBounceVector.z *= -1;

    }

    return randomBounceVector;
}

__device__ float* handleRayIntersection(float3 rayOrigin, float3 rayVector, float**worldObjectData, int numWorldObjects, bool* intersected, float* shortestIntersectionDistance) {
    float* intersectionObject;
    for(int i = 0; i < numWorldObjects; i++) {
        switch(static_cast<int>(worldObjectData[i][0])) {
            case 0:
                float sphereVecX = worldObjectData[i][6] - rayOrigin.x;
                float sphereVecY = worldObjectData[i][7] - rayOrigin.y;
                float sphereVecZ = worldObjectData[i][8] - rayOrigin.z;
                float sphereRadius = worldObjectData[i][9];
                float dotProduct = (sphereVecX*rayVector.x + sphereVecY*rayVector.y + sphereVecZ*rayVector.z); //LENGTH t1c
                if(dotProduct<0){break;}
                float distanceToSphereCenter = sqrtf(sphereVecX*sphereVecX + sphereVecY*sphereVecY + sphereVecZ*sphereVecZ);
                float rayPerpendicularDistanceToCenter = sqrtf(distanceToSphereCenter*distanceToSphereCenter - dotProduct*dotProduct);
                if(rayPerpendicularDistanceToCenter > sphereRadius) {break;}
                float distanceToSphereEdge = sqrtf(sphereRadius*sphereRadius - rayPerpendicularDistanceToCenter*rayPerpendicularDistanceToCenter);
                float intersectionDistance1 = dotProduct - distanceToSphereEdge;
                float intersectionDistance2 = dotProduct + distanceToSphereEdge;
                if(intersectionDistance1 > 0 ){
                    if(!*intersected) {
                        *intersected = true;
                        *shortestIntersectionDistance = intersectionDistance1;
                        intersectionObject = worldObjectData[i];
                    } else if(intersectionDistance1 < *shortestIntersectionDistance){
                        *intersected = true;
                        *shortestIntersectionDistance = intersectionDistance1;
                        intersectionObject = worldObjectData[i];
                    }
                }
                // if(!*intersected) {
                //     if(intersectionDistance1 > 0) {
                //         *shortestIntersectionDistance = intersectionDistance1;
                //         intersectionObject = worldObjectData[i];
                //         *intersected = true;
                //     }
                //     if(intersectionDistance2 > 0 && intersectionDistance2 < intersectionDistance1) {
                //         *shortestIntersectionDistance = intersectionDistance2;
                //         intersectionObject = worldObjectData[i];
                //         *intersected = true;
                //     }
                // } else{
                //     if(intersectionDistance1 > 0 && intersectionDistance1 < *shortestIntersectionDistance) {
                //         *shortestIntersectionDistance = intersectionDistance1;
                //         intersectionObject = worldObjectData[i];
                //     }
                //     if(intersectionDistance2 > 0 && intersectionDistance2 < *shortestIntersectionDistance) {
                //         *shortestIntersectionDistance = intersectionDistance2;
                //         intersectionObject = worldObjectData[i];
                //     }
                // }
                break;
            
            case 1:
                float distanceToIntersection = 
                (worldObjectData[i][18]-worldObjectData[i][15]*rayOrigin.x-worldObjectData[i][16]*rayOrigin.y-worldObjectData[i][17]*rayOrigin.z)
                /(worldObjectData[i][15]*rayVector.x + worldObjectData[i][16]*rayVector.y + worldObjectData[i][17]*rayVector.z);
                if(distanceToIntersection < 0 || isnan(distanceToIntersection)) {break;}
                //Determine if point is in triangle
                float3 intersectionPoint = make_float3(
                    rayOrigin.x+rayVector.x*distanceToIntersection, 
                    rayOrigin.y+rayVector.y*distanceToIntersection, 
                    rayOrigin.z+rayVector.z*distanceToIntersection
                );

                float3 vecA = make_float3(
                    intersectionPoint.x-worldObjectData[i][6],
                    intersectionPoint.y-worldObjectData[i][7],
                    intersectionPoint.z-worldObjectData[i][8]
                );    
                float triangle1Area=computeCrossProductLength(vecA,make_float3(worldObjectData[i][19],worldObjectData[i][20],worldObjectData[i][21]));
                float triangle2Area=computeCrossProductLength(vecA,make_float3(worldObjectData[i][22],worldObjectData[i][23],worldObjectData[i][24]));
                float triangle3Area=computeCrossProductLength(make_float3(intersectionPoint.x-worldObjectData[i][9],intersectionPoint.y-worldObjectData[i][10],intersectionPoint.z-worldObjectData[i][11]),make_float3(worldObjectData[i][25],worldObjectData[i][26],worldObjectData[i][27]));
                if(fabs(triangle1Area+triangle2Area+triangle3Area-worldObjectData[i][28]) > 0.0001) {
                    break;
                }
                if(!*intersected && distanceToIntersection>0) {
                    *shortestIntersectionDistance = distanceToIntersection;
                    intersectionObject = worldObjectData[i];
                    *intersected = true;
                } else if(distanceToIntersection>0 && distanceToIntersection < *shortestIntersectionDistance) {
                    *shortestIntersectionDistance = distanceToIntersection;
                    intersectionObject = worldObjectData[i];
                }
                break;
        }
    }
    return intersectionObject;
}


//Alternate Method Of Handling Intersection by Avoiding the Use of Bool Value (Was useful for troubleshooting!)
__device__ float* handleRayIntersection2(float3 rayOrigin, float3 rayVector, float**worldObjectData, int numWorldObjects, float* shortestIntersectionDistance,float* troubleShootData,int x, int y) {
    float* intersectionObject;
    for(int i = 0; i < numWorldObjects; i++) {
        switch(static_cast<int>(worldObjectData[i][0])) {
            case 0:
                float sphereVecX = worldObjectData[i][6] - rayOrigin.x;
                float sphereVecY = worldObjectData[i][7] - rayOrigin.y;
                float sphereVecZ = worldObjectData[i][8] - rayOrigin.z;
                float sphereRadius = worldObjectData[i][9];
                float dotProduct = (sphereVecX*rayVector.x + sphereVecY*rayVector.y + sphereVecZ*rayVector.z); //LENGTH t1c
                if(dotProduct<0){break;}
                float distanceToSphereCenter = sqrtf(sphereVecX*sphereVecX + sphereVecY*sphereVecY + sphereVecZ*sphereVecZ);
                float rayPerpendicularDistanceToCenter = sqrtf(distanceToSphereCenter*distanceToSphereCenter - dotProduct*dotProduct);
                if(rayPerpendicularDistanceToCenter > sphereRadius) {break;}
                float distanceToSphereEdge = sqrtf(sphereRadius*sphereRadius - rayPerpendicularDistanceToCenter*rayPerpendicularDistanceToCenter);
                float intersectionDistance1 = dotProduct - distanceToSphereEdge;
                float intersectionDistance2 = dotProduct + distanceToSphereEdge;
                if(intersectionDistance1>0 && intersectionDistance1 < *shortestIntersectionDistance) {
                    *shortestIntersectionDistance = intersectionDistance1;
                    intersectionObject = worldObjectData[i];
                }
                if(intersectionDistance2>0 && intersectionDistance2 < *shortestIntersectionDistance) {
                    *shortestIntersectionDistance = intersectionDistance2;
                    intersectionObject = worldObjectData[i];
                }
                break;
            
            case 1:
                float distanceToIntersection = 
                (worldObjectData[i][18]-worldObjectData[i][15]*rayOrigin.x-worldObjectData[i][16]*rayOrigin.y-worldObjectData[i][17]*rayOrigin.z)
                /(worldObjectData[i][15]*rayVector.x + worldObjectData[i][16]*rayVector.y + worldObjectData[i][17]*rayVector.z);
                if(distanceToIntersection < 0 || isnan(distanceToIntersection)) {break;}
                //Determine if point is in triangle
                float3 intersectionPoint = make_float3(
                    rayOrigin.x+rayVector.x*distanceToIntersection, 
                    rayOrigin.y+rayVector.y*distanceToIntersection, 
                    rayOrigin.z+rayVector.z*distanceToIntersection
                );

                float3 vecA = make_float3(
                    intersectionPoint.x-worldObjectData[i][6],
                    intersectionPoint.y-worldObjectData[i][7],
                    intersectionPoint.z-worldObjectData[i][8]
                );    
                float triangle1Area=computeCrossProductLength(vecA,make_float3(worldObjectData[i][19],worldObjectData[i][20],worldObjectData[i][21]));
                float triangle2Area=computeCrossProductLength(vecA,make_float3(worldObjectData[i][22],worldObjectData[i][23],worldObjectData[i][24]));
                float triangle3Area=computeCrossProductLength(make_float3(intersectionPoint.x-worldObjectData[i][9],intersectionPoint.y-worldObjectData[i][10],intersectionPoint.z-worldObjectData[i][11]),make_float3(worldObjectData[i][25],worldObjectData[i][26],worldObjectData[i][27]));
                if(fabs(triangle1Area+triangle2Area+triangle3Area-worldObjectData[i][28]) > 0.0001) {
                    break;
                }
                if(distanceToIntersection>0 && distanceToIntersection<*shortestIntersectionDistance) {
                    *shortestIntersectionDistance = distanceToIntersection;
                    intersectionObject = worldObjectData[i];
                }
                break;
        }
    }
    if(!intersectionObject) {
        intersectionObject = new float[30];
        intersectionObject[0] = -1;
    }
    if(x==600 && y==400) {
        troubleShootData[0] = *shortestIntersectionDistance;
        troubleShootData[1] = intersectionObject[1];
        troubleShootData[2] = intersectionObject[2];
        troubleShootData[3] = intersectionObject[3];
        troubleShootData[4] = intersectionObject[5];
        troubleShootData[5] = intersectionObject[6];
        troubleShootData[6] = intersectionObject[7];
        troubleShootData[7] = intersectionObject[8];
    }
    return intersectionObject;
}

__device__ float3 getNormalVector(float* intersectionObject, float3 intersectionPoint, float3 rayVector) {
    float3 normalVector;
    switch(static_cast<int>(intersectionObject[0])) {
        case 0:
            normalVector.x = intersectionPoint.x-intersectionObject[6];
            normalVector.y = intersectionPoint.y-intersectionObject[7];
            normalVector.z = intersectionPoint.z-intersectionObject[8];
            break;

        case 1:
            normalVector.x = intersectionObject[15];
            normalVector.y = intersectionObject[16];
            normalVector.z = intersectionObject[17];
            break;
    }
    if((normalVector.x*rayVector.x + normalVector.y*rayVector.y + normalVector.z*rayVector.z)>0) {
        normalVector.x*=-1;
        normalVector.y*=-1;
        normalVector.z*=-1;
    }
    //NORMALIZE VECTOR:
    float inverseLength = rsqrtf(normalVector.x*normalVector.x + normalVector.y*normalVector.y + normalVector.z*normalVector.z);
    normalVector.x *= inverseLength;
    normalVector.y *= inverseLength;
    normalVector.z *= inverseLength;
    return normalVector;

}

__global__ void RayTracingKernelSimple(unsigned int seed, int imageWidth, int imageHeight,int numWorldObjects, float* cameraData, unsigned char* imageData, float** worldObjectData,float* troubleShootData) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < imageWidth && y < imageHeight) {
        int pixelIndex = (y * imageWidth + x) * 3;
        
        hiprandState randomGenerator;
        hiprand_init(seed+x+y*imageWidth,0,0,&randomGenerator);
        //Rotate Vector:
        float3 rayVector = getVector(x,y,imageWidth,imageHeight,cameraData[4],cameraData[5],cameraData[6],cameraData[7],cameraData[3]);
        float3 rayOrigin = make_float3(cameraData[0],cameraData[1],cameraData[2]);
        float3 color = make_float3(1,1,1);
        float3 intersectionPoint;
        float3 bounceVec;
        float3 normVector;
        float shortestIntersectionDistance;

        shortestIntersectionDistance = 999999;
        float* intersectionObject = handleRayIntersection2(rayOrigin,rayVector,worldObjectData,numWorldObjects,&shortestIntersectionDistance,troubleShootData,x,y);
        
        if(shortestIntersectionDistance < 9999){
            color.x*=intersectionObject[1];
            color.y*=intersectionObject[2];
            color.z*=intersectionObject[3];
            intersectionPoint = make_float3(
            rayOrigin.x + rayVector.x*shortestIntersectionDistance,
            rayOrigin.y + rayVector.y*shortestIntersectionDistance,
            rayOrigin.z + rayVector.z*shortestIntersectionDistance
            );
            normVector = getNormalVector(intersectionObject,intersectionPoint,rayVector);
            bounceVec = getRandomBounceVector(normVector,&randomGenerator);
            if(x==600&&y==400){
                troubleShootData[0] = normVector.x;
                troubleShootData[1] = normVector.y;
                troubleShootData[2] = normVector.z;
                troubleShootData[3] = bounceVec.x;
                troubleShootData[4] = bounceVec.y;
                troubleShootData[5] = bounceVec.z;
            }
        } else{
            float3 skyColor = getSkyBoxColor(rayVector,0.75);
            color.x *= skyColor.x;
            color.y *= skyColor.y;
            color.z *= skyColor.z;
            imageData[pixelIndex] = static_cast<unsigned char>(255.0f * color.x); // Red
            imageData[pixelIndex + 1] = static_cast<unsigned char>(255.0f * color.y); // Green
            imageData[pixelIndex + 2] = static_cast<unsigned char>(255.0f * color.z); // Blue
            return;
        }
        imageData[pixelIndex] = static_cast<unsigned char>(255.0f * color.x); // Red
        imageData[pixelIndex + 1] = static_cast<unsigned char>(255.0f * color.y); // Green
        imageData[pixelIndex + 2] = static_cast<unsigned char>(255.0f * color.z); 
        
    }
};





__global__ void RayTracingKernel(unsigned int seed, int frame, int imageWidth, int imageHeight,int numWorldObjects, float* cameraData, unsigned char* imageData, float** worldObjectData,float* troubleShootData) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < imageWidth && y < imageHeight) {
        int pixelIndex = (y * imageWidth + x) * 3;
        
        hiprandState randomGenerator;
        hiprand_init(seed+x+y*imageWidth+frame*79231,0,0,&randomGenerator);
        //Rotate Vector:
        float3 rayVector = getVector(x,y,imageWidth,imageHeight,cameraData[4],cameraData[5],cameraData[6],cameraData[7],cameraData[3]);
        float3 rayOrigin = make_float3(cameraData[0],cameraData[1],cameraData[2]);
        float3 color = make_float3(1,1,1);
        float3 lightColor = make_float3(0.0f,0.0f,0.0f);
        float3 normalVector;
        bool intersected;
        float shortestIntersectionDistance;
        float* intersectionObject;
        for(int i =0; i < 10; i ++) {
            intersected = false;
            shortestIntersectionDistance = 999999999999;
            intersectionObject = handleRayIntersection(rayOrigin,rayVector,worldObjectData,numWorldObjects,&intersected,&shortestIntersectionDistance);
            if(intersected){
                if(intersectionObject[5] > 0.5) {
                    lightColor.x = intersectionObject[1];
                    lightColor.y = intersectionObject[2];
                    lightColor.z = intersectionObject[3];
                    break;
                } else {
                    color.x*=intersectionObject[1];
                    color.y*=intersectionObject[2];
                    color.z*=intersectionObject[3];
                    rayOrigin.x += rayVector.x*shortestIntersectionDistance;
                    rayOrigin.y += rayVector.y*shortestIntersectionDistance;
                    rayOrigin.z += rayVector.z*shortestIntersectionDistance;
                    normalVector = getNormalVector(intersectionObject,rayOrigin,rayVector);
                    rayOrigin.x+=normalVector.x*0.000001;
                    rayOrigin.y+=normalVector.y*0.000001;
                    rayOrigin.z+=normalVector.z*0.000001;
                    rayVector = getRandomBounceVector(normalVector,&randomGenerator);
                }
            }
        }
        color.x *= lightColor.x;
        color.y *= lightColor.y;
        color.z *= lightColor.z;
        
        float proportionalConstant = 1.0/static_cast<float>(frame);
        if(x==600 && y==400) {
            troubleShootData[0] = lightColor.x;
            troubleShootData[1] = lightColor.x;
            troubleShootData[2] = lightColor.x;
            troubleShootData[3] = intersectionObject[5];
            troubleShootData[4] = color.x;
            troubleShootData[5] = color.x;
            troubleShootData[6] = color.x;
            troubleShootData[7] = proportionalConstant;
            troubleShootData[8] = frame;

        }
        imageData[pixelIndex] = imageData[pixelIndex] * (1-proportionalConstant) + proportionalConstant*static_cast<unsigned char>(255.0f * color.x); // Red
        imageData[pixelIndex + 1] = imageData[pixelIndex+1] * (1-proportionalConstant) + proportionalConstant*static_cast<unsigned char>(255.0f * color.y); // Green
        imageData[pixelIndex + 2] = imageData[pixelIndex+2] * (1-proportionalConstant) + proportionalConstant*static_cast<unsigned char>(255.0f * color.z);
    }
};

__global__ void DiffuseRender(unsigned int seed, int frame,int maxRayBounces, int imageWidth, int imageHeight,int numWorldObjects, float* cameraData, float* imageDataArray, float** worldObjectData,float* troubleShootData) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < imageWidth && y < imageHeight) {
        int pixelIndex = (y * imageWidth + x) * 3;
        
        hiprandState randomGenerator;
        hiprand_init(seed+frame*79231,x+y*imageWidth,0,&randomGenerator);
        //Rotate Vector:
        float3 rayVector = getVector(x,y,imageWidth,imageHeight,cameraData[4],cameraData[5],cameraData[6],cameraData[7],cameraData[3]);
        float3 rayOrigin = make_float3(cameraData[0],cameraData[1],cameraData[2]);
        float3 color = make_float3(1,1,1);
        float3 lightColor = make_float3(0.1f,0.1f,0.1f);
        float3 normalVector;
        bool intersected;
        float shortestIntersectionDistance;
        float* intersectionObject;
        for(int i =0; i < maxRayBounces; i ++) {
            intersected = false;
            shortestIntersectionDistance = 999999999999;
            intersectionObject = handleRayIntersection(rayOrigin,rayVector,worldObjectData,numWorldObjects,&intersected,&shortestIntersectionDistance);
            if(intersected){
                if(intersectionObject[5] > 0.5) {
                    lightColor.x = intersectionObject[1];
                    lightColor.y = intersectionObject[2];
                    lightColor.z = intersectionObject[3];
                    break;
                } else {
                    color.x*=intersectionObject[1];
                    color.y*=intersectionObject[2];
                    color.z*=intersectionObject[3];
                    rayOrigin.x += rayVector.x*shortestIntersectionDistance;
                    rayOrigin.y += rayVector.y*shortestIntersectionDistance;
                    rayOrigin.z += rayVector.z*shortestIntersectionDistance;
                    normalVector = getNormalVector(intersectionObject,rayOrigin,rayVector);
                    rayOrigin.x+=normalVector.x*0.000001;
                    rayOrigin.y+=normalVector.y*0.000001;
                    rayOrigin.z+=normalVector.z*0.000001;
                    rayVector = getRandomBounceVector(normalVector,&randomGenerator);
                }
            }
        }
        color.x *= lightColor.x;
        color.y *= lightColor.y;
        color.z *= lightColor.z;
        
        imageDataArray[pixelIndex] = imageDataArray[pixelIndex] + color.x; // Red
        imageDataArray[pixelIndex + 1] = imageDataArray[pixelIndex+1] + color.y; // Green
        imageDataArray[pixelIndex + 2] = imageDataArray[pixelIndex+2] + color.z;

        if(x==600 && y==400) {
            troubleShootData[0] = imageDataArray[pixelIndex];
            troubleShootData[1] = imageDataArray[pixelIndex+1];
            troubleShootData[2] = imageDataArray[pixelIndex+2];
            troubleShootData[3] = color.x;
            troubleShootData[4] = color.y;
            troubleShootData[5] = color.z;
        }
    }
};


__global__ void updateImage(int frame, int imageWidth, int imageHeight,float* imageDataArray,unsigned char* imageData,float* troubleShootData) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < imageWidth && y < imageHeight) {
        int pixelIndex = (y * imageWidth + x) * 3;
        
        
        imageData[pixelIndex] = static_cast<unsigned char>(255*fminf(1,static_cast<float>(imageDataArray[pixelIndex]/(0.2*static_cast<float>(frame))))); // Red
        imageData[pixelIndex + 1] = static_cast<unsigned char>(255*fminf(1,static_cast<float>(imageDataArray[pixelIndex+1]/(0.2*static_cast<float>(frame))))); // Green
        imageData[pixelIndex + 2] = static_cast<unsigned char>(255*fminf(1,static_cast<float>(imageDataArray[pixelIndex+2]/(0.2*static_cast<float>(frame)))));
        if(x==600 && y==400) {
            troubleShootData[0] = imageData[pixelIndex];
            troubleShootData[1] = imageData[pixelIndex+1];
            troubleShootData[2] = imageData[pixelIndex+2];
            troubleShootData[3] = imageDataArray[pixelIndex];
            troubleShootData[4] = imageDataArray[pixelIndex+1];
            troubleShootData[5] = imageDataArray[pixelIndex+2];
        }
    }
};





Display::Display(int displayWidth,int displayHeight,Camera* viewCamera): displayWidth(displayWidth),displayHeight(displayHeight),viewCamera(viewCamera),displayState(MAIN_MENU) {
    this->imageData = new unsigned char[this->displayWidth * this->displayHeight * 3];
    std::cout<<"Display Created"<<std::endl;

    //Initializes SDL VIDEO
    if (SDL_Init(SDL_INIT_VIDEO) != 0 || TTF_Init() == -1) {
        std::cout<<"MAJOR ERROR: FAILURE TO LOAD SDL"<<std::endl;
    }

    //Make Display Window
    this->displayWindow = SDL_CreateWindow("RayTracing", SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED, this->displayWidth, this->displayHeight, SDL_WINDOW_RESIZABLE);
    if (this->displayWindow==NULL) {
        std::cout<<"MAJOR ERROR: FAILURE TO CREATE SDL WINDOW"<<std::endl;
    }
    this->resetCursor();
    
    //Initialize and Create SDL Renderer to display image
    this->imageRenderer = SDL_CreateRenderer(this->displayWindow, -1, SDL_RENDERER_ACCELERATED | SDL_RENDERER_PRESENTVSYNC);
    SDL_SetRenderDrawBlendMode(this->imageRenderer, SDL_BLENDMODE_BLEND);
    
    this->imageTexture = SDL_CreateTexture(this->imageRenderer, SDL_PIXELFORMAT_RGB24, SDL_TEXTUREACCESS_STREAMING, this->displayWidth, this->displayHeight);

    this->allocateGPUMemory();
    this->recieveGpuTroubleShootData = new float[30];
    std::random_device rd;
    this->randomSeedGenerator.seed(rd());
    this->seedDistribution = std::uniform_real_distribution<float>(0,1);
};



Display::~Display() {
    std::cout<<"Display Deconstructing"<<std::endl;

    if(this->gpuImage)
        hipFree(this->gpuImage);
    if(this->gpuCameraData)
        hipFree(this->gpuCameraData);
    if(this->displayWindow)
        SDL_DestroyWindow(this->displayWindow);
    if(this->imageTexture)
        SDL_DestroyTexture(this->imageTexture);
}




void Display::updateDisplayParameters(Settings* settings) {
    this->displayWidth = settings->getGraphicsSettingsValue("ScreenResolutionX");
    this->displayHeight = settings->getGraphicsSettingsValue("ScreenResolutionY");

}



#pragma region GPU MEMORY
void Display::allocateWorldData(World* world) {
    float** worldDataAsArray = world->getWorldObjectDataAsArray();
    this->numOfWorldObjects = world->numOfObjects;
    hipMalloc(&this->gpuWorldObjectData, this->numOfWorldObjects * sizeof(float*));
    for(size_t i = 0; i < this->numOfWorldObjects; i++) {
        float* objectData;
        hipMalloc(&objectData, 30 * sizeof(float));
        hipMemcpy(objectData, worldDataAsArray[i],30*sizeof(float),hipMemcpyHostToDevice);
        hipMemcpy(&this->gpuWorldObjectData[i],&objectData,sizeof(float*),hipMemcpyHostToDevice);
    }
}



void Display::allocateGPUMemory() {
    hipMalloc(&this->gpuTroubleShootData, 30*sizeof(float));
    hipMalloc(&this->gpuCameraData, 8 * sizeof(float));
    hipMalloc(&this->gpuImage, this->displayWidth * this->displayHeight * 3 * sizeof(unsigned char));
}

void Display::reAllocateGPUMemory() {
    hipFree(this->gpuImage);
    hipMalloc(&this->gpuCameraData, 8 * sizeof(float));
    hipMalloc(&this->gpuImage, this->displayWidth * this->displayHeight * 3 * sizeof(unsigned char));
}

void Display::updateGPUData() {
    hipMemcpy(this->gpuCameraData, this->viewCamera->cameraData, 8 * sizeof(float), hipMemcpyHostToDevice);
}

void Display::copyImageData() {
    hipMemcpy(this->gpuImage, this->imageData, this->displayWidth * this->displayHeight * 3 * sizeof(unsigned char),hipMemcpyHostToDevice);
}
#pragma endregion




void Display::getPixelInfo(int x, int y) {
    int index = (y * this->displayWidth + x) * 3;
    std::cout << "Pixel (" << x << ", " << y << "): ( " << static_cast<int>(this->imageData[index]) << ", " << static_cast<int>(this->imageData[index+1]) << ", " << static_cast<int>(this->imageData[index+2]) << " )" << std::endl;
}

void Display::resetCursor() {
    SDL_WarpMouseInWindow(this->displayWindow, this->displayWidth / 2, this->displayHeight / 2);
    this->previousMousePosX = this->displayWidth/2;
    this->previousMousePosY = this->displayHeight/2;
    this->mousePosX = this->displayWidth/2;
    this->mousePosY = this->displayHeight/2;
}

/*
NEXT STEP IS TO REORGANIZE EVENTS INTO THE EVENT MANAGER CLASS
NEEDED QUITE BADLY

*/




void Display::renderScene() {
    this->renderImage();
    SDL_RenderClear(this->imageRenderer);
    SDL_UpdateTexture(this->imageTexture, NULL, this->imageData, this->displayWidth*3);
    SDL_RenderCopy(this->imageRenderer, this->imageTexture, NULL, NULL);
    SDL_RenderPresent(this->imageRenderer);
}


void Display::handleDisplaySizeChange(int newDisplayWidth, int newDisplayHeight) {
    this->displayWidth = newDisplayWidth;
    this->displayHeight = newDisplayHeight;
    delete[] this->imageData;
    this->imageData = new unsigned char[this->displayWidth * this->displayHeight * 3];
    SDL_DestroyTexture(this->imageTexture);
    this->imageTexture = SDL_CreateTexture(this->imageRenderer, SDL_PIXELFORMAT_RGB24, SDL_TEXTUREACCESS_STREAMING, this->displayWidth, this->displayHeight);
    hipFree(this->gpuImage);
    hipMalloc(&this->gpuImage, this->displayWidth * this->displayHeight * 3 * sizeof(unsigned char));

}







void Display::renderImage() {
    dim3 threads(16, 8);
    dim3 blocksPerGrid(ceil(displayWidth / static_cast<float>(threads.x)),
                        ceil(displayHeight / static_cast<float>(threads.y)));

    //Update GPU Data
    this->updateGPUData();

    int randomGenSeed = static_cast<int>(99999.0*this->seedDistribution(this->randomSeedGenerator));
    

    // Launch the CUDA kernel defined inside the class
    RayTracingKernel<<<blocksPerGrid, threads>>>(randomGenSeed, 1, displayWidth, displayHeight, this->numOfWorldObjects,this->gpuCameraData,this->gpuImage, this->gpuWorldObjectData,this->gpuTroubleShootData);

    hipError_t kernelError = hipGetLastError();
    if (kernelError != hipSuccess) {
        std::cout << "CUDA kernel launch error: " << hipGetErrorString(kernelError) << std::endl;
        hipFree(this->gpuImage); // Free allocated memory
        return;
    }

    // Wait for kernel to finish
    hipDeviceSynchronize();

    // Copy the processed image back from GPU to CPU
    hipMemcpy(this->imageData, this->gpuImage, displayWidth * displayHeight * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);

    //USED TO COPY TROUBLESHOOT DATA TO CPU
    // hipMemcpy(this->recieveGpuTroubleShootData, this->gpuTroubleShootData,30*sizeof(float),hipMemcpyDeviceToHost);
    // std::cout<<"----------------------"<<std::endl;
    // for(size_t i = 0; i<8; i++) {
    //     std::cout<<"Data Index: "<<i<<"   Data Value: "<<this->recieveGpuTroubleShootData[i]<<std::endl;
    // }
    // Free GPU memory
};


void Display::seriousRenderOfImage(int numOfFrames,int maxRayBounces) {
    dim3 threads(16, 8);
    dim3 blocksPerGrid(ceil(displayWidth / static_cast<float>(threads.x)),
                        ceil(displayHeight / static_cast<float>(threads.y)));

    float* imageDataArray = new float[this->displayWidth*this->displayHeight*3];
    float* gpuImageDataArray;
    hipMalloc(&gpuImageDataArray,this->displayWidth*this->displayHeight*3*sizeof(float));

    for(int frame = 1; frame<=numOfFrames; frame++) {
        std::cout<<"Rendering Frame: "<<frame<<std::endl;
        //Update GPU Data
        this->updateGPUData();
        this->copyImageData();

        int randomGenSeed = static_cast<int>(99999.0*this->seedDistribution(this->randomSeedGenerator));
        
        
        // Launch the CUDA kernel defined inside the class
        DiffuseRender<<<blocksPerGrid, threads>>>(randomGenSeed,frame,maxRayBounces,displayWidth, displayHeight, this->numOfWorldObjects,this->gpuCameraData,gpuImageDataArray, this->gpuWorldObjectData,this->gpuTroubleShootData);
        
        hipError_t kernelError = hipGetLastError();
        if (kernelError != hipSuccess) {
            std::cout << "CUDA kernel launch error: " << hipGetErrorString(kernelError) << std::endl;
            hipFree(this->gpuImage); // Free allocated memory
            return;
        }

        // Wait for kernel to finish
        hipDeviceSynchronize();

        updateImage<<<blocksPerGrid, threads>>>(frame,displayWidth, displayHeight,gpuImageDataArray,this->gpuImage,this->gpuTroubleShootData);
        kernelError = hipGetLastError();
        if (kernelError != hipSuccess) {
            std::cout << "CUDA kernel launch error: " << hipGetErrorString(kernelError) << std::endl;
            hipFree(this->gpuImage); // Free allocated memory
            return;
        }

        // Wait for kernel to finish
        hipDeviceSynchronize();

        // Copy the processed image back from GPU to CPU
        hipMemcpy(this->imageData, this->gpuImage, displayWidth * displayHeight * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);

        //Updates DISPLAYED Image
        SDL_RenderClear(this->imageRenderer);
        SDL_UpdateTexture(this->imageTexture, NULL, this->imageData, this->displayWidth*3);
        SDL_RenderCopy(this->imageRenderer, this->imageTexture, NULL, NULL);
        SDL_RenderPresent(this->imageRenderer);
    }
}



void Display::customResolutionRender(std::string fileName, int numOfFrames,int maxRayBounces, int resolutionX, int resolutionY) {
    dim3 threads(16, 8);
    dim3 blocksPerGrid(ceil(resolutionX / static_cast<float>(threads.x)),
                        ceil(resolutionY / static_cast<float>(threads.y)));

    float* imageDataArray = new float[resolutionX*resolutionY*3];
    float* gpuImageDataArray;
    hipMalloc(&gpuImageDataArray,resolutionX*resolutionY*3*sizeof(float));

    unsigned char* customResImageData = new unsigned char[resolutionX*resolutionY*3];
    unsigned char* customResGPUImageData;
    hipMalloc(&customResGPUImageData,resolutionX*resolutionY*3*sizeof(unsigned char));

    hipError_t kernelError;

    for(int frame = 1; frame<=numOfFrames; frame++) {
        std::cout<<"Rendering Frame: "<<frame<<std::endl;
        //Update GPU Data
        this->updateGPUData();
        this->copyImageData();

        int randomGenSeed = static_cast<int>(99999.0*this->seedDistribution(this->randomSeedGenerator));
        
        
        // Launch the CUDA kernel defined inside the class
        DiffuseRender<<<blocksPerGrid, threads>>>(randomGenSeed,frame,maxRayBounces,resolutionX, resolutionY, this->numOfWorldObjects,this->gpuCameraData,gpuImageDataArray, this->gpuWorldObjectData,this->gpuTroubleShootData);
        
        kernelError = hipGetLastError();
        if (kernelError != hipSuccess) {
            std::cout << "CUDA kernel launch error: " << hipGetErrorString(kernelError) << std::endl;
            hipFree(this->gpuImage); // Free allocated memory
            return;
        }

        // Wait for kernel to finish
        hipDeviceSynchronize();
    }
    updateImage<<<blocksPerGrid, threads>>>(numOfFrames,resolutionX, resolutionY,gpuImageDataArray,customResGPUImageData,this->gpuTroubleShootData);
        kernelError = hipGetLastError();
        if (kernelError != hipSuccess) {
            std::cout << "CUDA kernel launch error: " << hipGetErrorString(kernelError) << std::endl;
            hipFree(this->gpuImage); // Free allocated memory
            return;
        }

    // Wait for kernel to finish
    hipDeviceSynchronize();

    // Copy the processed image back from GPU to CPU
    hipMemcpy(customResImageData, customResGPUImageData, resolutionX*resolutionY * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);

    std::cout<<"saving image..."<<std::endl;
    // Write image data to BMP file
    SDL_Surface* surface = SDL_CreateRGBSurfaceFrom(customResImageData, resolutionX, resolutionY, 24, resolutionX * 3, 0xFF0000, 0x00FF00, 0x0000FF, 0);
    std::string filePath = "./Saved Images/" +fileName + ".bmp";
    std::cout<<"FilePath: "<<filePath<<std::endl;
    SDL_SaveBMP(surface,filePath.c_str());
}


void Display::saveImage(const std::string& fileName) {
    std::cout<<"saving image..."<<std::endl;
    // Write image data to BMP file
    SDL_Surface* surface = SDL_CreateRGBSurface(0, displayWidth, displayHeight, 32, 0, 0, 0, 0);
    SDL_RenderReadPixels(this->imageRenderer,NULL,SDL_PIXELFORMAT_ARGB8888, surface->pixels, surface->pitch);

    std::string filePath = "./Saved Images/" +fileName + ".bmp";
    std::cout<<"FilePath: "<<filePath<<std::endl;
    SDL_SaveBMP(surface,filePath.c_str());
    
}

